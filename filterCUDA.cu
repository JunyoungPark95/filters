#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "config.h"
#include "filterCUDA.h"

#include <time.h>

#define MAX(a, b) a > b ? a : b
#define MIN(a, b) a < b ? a : b 

__global__ void kernel_box_filter(float *estColor, float *inputColor, int xres, int yres, float *time) {
	clock_t start, end;
	double result;

	start = clock(); // running time evaluation start
	const int cx = blockDim.x * blockIdx.x + threadIdx.x;
	const int cy = blockDim.y * blockIdx.y + threadIdx.y;

	if (cx >= xres || cy >= yres) {
		return;
	}
	
	int index = cy * xres + cx;

	int sampleCount = 0;
	float color[3] = { 0.0f, };
	int sx = MAX(cx - KernelRadius, 0); // adaptively controll the size of kernel entirely 4 cases
	int sy = MAX(cy - KernelRadius, 0);
	int ex = MIN(cx + KernelRadius, xres - 1);
	int ey = MIN(cy + KernelRadius, yres - 1);
	for (int ny = sy; ny <= ey; ++ny) { // kernel
		for (int nx = sx; nx <= ex; ++nx) {
			color[0] += inputColor[3 * (xres * ny + nx) + 0];
			color[1] += inputColor[3 * (xres * ny + nx) + 1];
			color[2] += inputColor[3 * (xres * ny + nx) + 2];
			sampleCount++;
		}
	}
	estColor[3 * index + 0] = 1.f / (float)sampleCount * color[0];
	estColor[3 * index + 1] = 1.f / (float)sampleCount * color[1];
	estColor[3 * index + 2] = 1.f / (float)sampleCount * color[2];

	end = clock();
	result = (float)(end - start);

	time[1] = (result) / CLOCKS_PER_SEC;
}

extern "C" void BoxFiltering(float *outputColor, float* inputColor, int xres, int yres, float *time) { 

	float invN = (1.0f / ((float)KernelSize * KernelSize));
	float *estColor = new float[yres * xres * 3]; // estColor = sum(inputColor * 1/ N)

	// Cuda mem allocation
	// Host mem cpy
	float *d_input_color, *d_output_color;

	const int memSize = sizeof(float) * 3 * xres * yres;

	hipMalloc((void **)&d_input_color, memSize);
	hipMalloc((void **)&d_output_color, memSize);

	hipMemcpy(&d_input_color[0], &inputColor[0], memSize, hipMemcpyHostToDevice);

	// kernel function 
	// for loop => parallel execution

	const int blockDim = 16;
	dim3 block(blockDim, blockDim);
	dim3 grid((xres + block.x - 1) / block.x, (yres + block.y - 1) / block.y);

	kernel_box_filter<<<grid, block>>>(d_output_color, d_input_color, xres, yres, time);

	hipMemcpy(&outputColor[0], &d_output_color[0], memSize, hipMemcpyDeviceToHost);

	hipFree(d_input_color);
	hipFree(d_output_color);
}